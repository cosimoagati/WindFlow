
#include <hip/hip_runtime.h>
#include <functional>
#include <iostream>
#include <cmath>

// This is an example to show how you can pass an arbitrary
// __device__ function to a CUDA kernel using multiple types, such
// as a function pointer or a lambda expression.


using namespace std;

// Doesn't work if host_kernel calls THIS function!
__host__ __device__ float
square(float x) { return x * x; }

// template<typename F_t>
// __device__ void
// perform_op(F_t f, float *x, float *y, const int n)
// {
// 	for (int i = 0; i < n; ++i)
// 		y[i] = f(x[i]);
// }

template<typename F_t>
__global__ void
host_kernel(float *x, float *y, const int n, F_t f)
{
	// Double numbers and then square them!
	// const auto double_num = [] __device__ (float x) { return 2 * x; };
	// perform_op<decltype(double_num)>(double_num, x, y, n);
	// perform_op<decltype(square)>(square, y, y, n);

	for (int i = 0; i < n; ++i)
		y[i] = f(x[i]);
}

// template<typename F_t>
// void
// call_kernel(F_t kernel, float *x, float *y, const int n)
// {
// 	kernel<<<1, 1>>>(x, y, n);
// }

int
main(void)
{
	int N = 1<<20;
	float *x, *y;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	for (int i = 0; i < N; i++)
		x[i] = static_cast<float>(i);
	// This lambda function, however, works!
	auto square = [] __device__ (float x) {return x * x; };
	host_kernel<decltype(square)><<<1, 1>>>(x, y, N, square);
	hipDeviceSynchronize();
	// call_kernel(host_kernel, x, y, N);
	// cudaDeviceSynchronize(); // Wait for GPU before accessing on host.

	for (int i = 0; i < 100; i++) // Only verify up to 99 for simplicity.
		cout << y[i] << "\n";
	hipFree(x); // Free memory
	hipFree(y);

	return 0;
}