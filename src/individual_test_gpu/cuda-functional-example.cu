
#include <hip/hip_runtime.h>
#include <functional>
#include <iostream>
#include <cmath>

// This is an example to show how you can pass an arbitrary
// __device__ function to a CUDA kernel using multiple types, such
// as a function pointer or a lambda expression.


using namespace std;

__device__ float
square(float x) { return x * x; }

template<typename F_t>
__device__ void
perform_op(F_t f, float *x, float *y, const int n)
{
	for (int i = 0; i < n; ++i)
		y[i] = f(x[i]);
}

__global__ void
kernel_run(float *x, float *y, const int n)
{
	// Double numbers and then square them!
	const auto double_num = [] __device__ (float x) { return 2 * x; };
	perform_op<decltype(double_num)>(double_num, x, y, n);
	perform_op<decltype(square)>(square, y, y, n);
}

int main(void)
{
	int N = 1<<20;
	float *x, *y;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	for (int i = 0; i < N; i++)
		x[i] = static_cast<float>(i);
	kernel_run<<<1, 1>>>(x, y, N);
	hipDeviceSynchronize(); // Wait for GPU before accessing on host.
	for (int i = 0; i < 100; i++) // Only verify up to 99 for simplicity.
		cout << y[i] << "\n";
	hipFree(x); // Free memory
	hipFree(y);

	return 0;
}